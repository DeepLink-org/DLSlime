#include "hip/hip_runtime.h"
#include "ops/utils.cuh"
#include "ops/launch.cuh"

#include <algorithm>
#include <cstdint>
#include <stdexcept>

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <torch/torch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>

namespace slime {

__global__ __launch_bounds__(1024, 1) void all_gather_ll_kernel(int8_t*  q_ptr,
                                                                int8_t** ipc_buffer_ptr,
                                                                int**    ipc_signal_ptr,
                                                                int32_t  max_bs,
                                                                int32_t  num_head,
                                                                int32_t  head_size,
                                                                int32_t  itemsize,
                                                                int32_t  world_size,
                                                                int32_t  rank)
{

    const int num_sms = std::min(128, max_bs * num_head);

    const int sm_id   = blockIdx.x;
    const int warp_id = threadIdx.x / 32;
    const int lane_id = deep_ep::get_lane_id();

    // Vectorize Optimization
    using vec_t        = int4;
    const int VEC_SIZE = sizeof(int4);

    int num_msg_per_warp     = head_size * itemsize;
    int num_vec_msg_per_warp = num_msg_per_warp / VEC_SIZE;

    const int q_idx_base = sm_id * head_size * itemsize;
    const int q_size     = max_bs * num_head * head_size * itemsize;
    // write q to buffer
    for (int q_idx = q_idx_base; q_idx < q_size; q_idx += num_sms * head_size * itemsize) {
        int8_t* q_ptr_for_write          = q_ptr + q_idx;
        vec_t*  vec_q_ptr_for_write      = reinterpret_cast<vec_t*>(q_ptr_for_write);
        int     buffer_idx               = q_idx + q_size * rank;
        int8_t* buffer_ptr_for_write     = ipc_buffer_ptr[warp_id] + buffer_idx;
        vec_t*  vec_buffer_ptr_for_write = reinterpret_cast<vec_t*>(buffer_ptr_for_write);

        UNROLLED_WARP_COPY(8,
                           lane_id,
                           num_vec_msg_per_warp,
                           vec_buffer_ptr_for_write,
                           vec_q_ptr_for_write,
                           deep_ep::ld_nc_global,
                           deep_ep::st_na_global);
    }
    __syncwarp();

    // Step 2. signal <= 1
    // check struggler
    // barrier or signal pingpong

    int* signal_ptr = ipc_signal_ptr[warp_id];
    if (lane_id == 0) {
        atomicAdd_system(signal_ptr + rank, 1);
    }

    __syncthreads();

    // Step 3. sync
    int* local_signal_buffer = ipc_signal_ptr[rank];
    if (threadIdx.x == 0 and blockIdx.x == 0) {
        for (int i = 0; i < world_size; ++i) {
            // load acquire atomic sys global
            while (__ldg(local_signal_buffer + i) < num_sms) {
                __threadfence_system();
            }
            local_signal_buffer[i] = 0;
        }
    }
    __syncthreads();

    // // Step 4. 复制 buffer[rank] 到 packed_buffer_ptr
    // // 计算总数据大小
    // const int total_data_size = q_size * world_size;    // 所有 rank 的总数据量
    // const int rank_data_size  = q_size;                 // 当前 rank 的数据量
    // const int rank_offset     = rank * rank_data_size;  // 当前 rank 在 buffer 中的偏移

    // // 使用向量加载/存储优化复制效率
    // int8_t* src_base = ipc_buffer_ptr[0] + rank_offset;  // buffer[rank] 的起始地址
    // int8_t* dst_base = packed_buffer_ptr + rank_offset;  // 目标地址（保持相同偏移）

    // // 计算需要复制的向量数量
    // const int total_vecs     = total_data_size / VEC_SIZE;
    // const int vecs_per_block = total_vecs / num_sms;
    // const int vec_start      = sm_id * vecs_per_block;
    // const int vec_end        = min((sm_id + 1) * vecs_per_block, total_vecs);

    // // 按块分配复制任务，每个 SM 负责一部分
    // vec_t* vec_src = reinterpret_cast<vec_t*>(src_base) + vec_start;
    // vec_t* vec_dst = reinterpret_cast<vec_t*>(dst_base) + vec_start;

    // // 线程级并行复制
    // for (int i = lane_id; i < (vec_end - vec_start); i += 32) {
    //     vec_dst[i] = deep_ep::ld_nc_global(&vec_src[i]);
    // }

    // __syncthreads();
}

void all_gather_ll(torch::Tensor q,
                            int8_t**      ipc_buffer_ptr,
                            int**         ipc_signal_ptr,
                            int32_t       max_bs,
                            int32_t       num_head,
                            int32_t       head_size,
                            int32_t       itemsize,
                            int32_t       world_size,
                            int32_t       rank)
{

    int8_t* q_ptr = reinterpret_cast<int8_t*>(q.data_ptr());

    int num_sms   = std::min(128, max_bs * num_head);
    int num_warps = world_size;

    int grid_dim  = num_sms;
    int block_dim = num_warps * 32;

    auto stream = at::cuda::getCurrentCUDAStream();
    SETUP_LAUNCH_CONFIG(grid_dim, block_dim, stream);
    LAUNCH_KERNEL(&cfg,
                  all_gather_ll_kernel,
                  q_ptr,
                  ipc_buffer_ptr,
                  ipc_signal_ptr,
                  max_bs,
                  num_head,
                  head_size,
                  itemsize,
                  world_size,
                  rank);

    // 检查内核错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    // return torch::empty({world_size, max_bs, num_head, head_size}, q.options().dtype(torch::kBFloat16));
}

}  // namespace slime
