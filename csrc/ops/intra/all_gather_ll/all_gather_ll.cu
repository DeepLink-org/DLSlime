#include "hip/hip_runtime.h"
#include "ops/utils.cuh"

#include <algorithm>

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace slime {

__global__ void all_gather_ll_kernel(int8_t*  q_ptr,
                                     int8_t** ipc_buffer_ptr,
                                     int**    ipc_signal_ptr,
                                     int32_t  max_bs,
                                     int32_t  num_head,
                                     int32_t  head_size,
                                     int32_t  itemsize,
                                     int32_t  world_size,
                                     int32_t  rank)
{
    const int num_sms              = 128;
    const int num_warps_per_sm     = 1;
    const int num_threads_per_warp = 32;

    const int num_sms_per_rank = num_sms / world_size;

    const int sm_id                = blockIdx.x;
    const int peer_rank_id         = sm_id / num_sms_per_rank;
    const int peer_rank_channel_id = sm_id % num_sms_per_rank;

    const int num_threads_per_channel = num_warps_per_sm * num_threads_per_warp;
    const int num_threads_per_rank    = num_sms_per_rank * num_warps_per_sm * num_threads_per_warp;
    const int num_total_msg_per_rank  = max_bs * num_head * head_size * itemsize;

    const int num_msg_per_thread = num_total_msg_per_rank / num_threads_per_rank;

    int8_t* buffer_ptr = ipc_buffer_ptr[peer_rank_id];
    int*    signal_ptr = ipc_signal_ptr[peer_rank_id];

    // Vectorize Optimization
    using vec_t                          = int4;
    const int VEC_SIZE                   = 16;
    const int num_vec_msg_per_thread     = num_msg_per_thread / VEC_SIZE;
    const int num_total_vec_msg_per_rank = num_total_msg_per_rank / VEC_SIZE;
    vec_t*    vec_buffer_ptr             = reinterpret_cast<vec_t*>(buffer_ptr);
    vec_t*    vec_q_ptr                  = reinterpret_cast<vec_t*>(q_ptr);

#pragma unroll 4
    for (int i = 0; i < num_vec_msg_per_thread; ++i) {
        // Step 1. Split q to num_sms_per_rank parts;
        int q_idx = peer_rank_channel_id * num_threads_per_channel * num_vec_msg_per_thread
                    + threadIdx.x * num_vec_msg_per_thread + i;
        int buffer_idx             = rank * num_total_vec_msg_per_rank + q_idx;
        vec_buffer_ptr[buffer_idx] = vec_q_ptr[q_idx];
    }

    __syncthreads();

    // Step 2. signal <= 1
    if (threadIdx.x == 0) {
        int signal_idx = rank;
        atomicAdd_system(signal_ptr + signal_idx, 1);
    }

    __syncthreads();

    // Step 3. sync
    int* local_signal_buffer = ipc_signal_ptr[rank];
    if (threadIdx.x == 0 and blockIdx.x == num_sms_per_rank * rank) {
        for (int i = 0; i < world_size; ++i) {
            while (__ldg(local_signal_buffer + i) < num_sms_per_rank) {
                __threadfence_system();
            }
            local_signal_buffer[i] = 0;
        }
    }
}

void all_gather_ll(uintptr_t q,
                   int8_t**  ipc_buffer_ptr,
                   int**     ipc_signal_ptr,
                   int32_t   max_bs,
                   int32_t   num_head,
                   int32_t   head_size,
                   int32_t   itemsize,
                   int32_t   world_size,
                   int32_t   rank)
{
    int num_sms     = 128;
    int num_warps   = 1;
    int num_threads = 32;

    int grid_dim  = num_sms;
    int block_dim = num_warps * num_threads;

    all_gather_ll_kernel<<<grid_dim, block_dim>>>(reinterpret_cast<int8_t*>(q),
                                                  ipc_buffer_ptr,
                                                  ipc_signal_ptr,
                                                  max_bs,
                                                  num_head,
                                                  head_size,
                                                  itemsize,
                                                  world_size,
                                                  rank);
}

}  // namespace slime
