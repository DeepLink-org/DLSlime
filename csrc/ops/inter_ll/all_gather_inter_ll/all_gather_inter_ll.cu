#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <stdexcept>

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <torch/torch.h>

#include "all_gather_inter_ll.h"
#include "ops/ibgda_device.cuh"
#include "ops/launch.cuh"
#include "ops/nvshmem_api.cuh"
#include "ops/utils.cuh"

namespace slime {

#define MAX_SMS 128

__device__ void coalescing_load(int8_t* src, int8_t* des, int length)
{
    size_t warp_id = threadIdx.x / 32;
    size_t lane_id = threadIdx.x % 32;

    // Step 1. Vectorized and Coalesced Data Copy
    constexpr int VEC_SIZE = 16;
    using vec_t            = int4;

    int global_thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
    int grid_stride       = gridDim.x * blockDim.x;

    size_t const vec_len = length / VEC_SIZE;
    vec_t*       vec_src = reinterpret_cast<vec_t*>(src);
    vec_t*       vec_des = reinterpret_cast<vec_t*>(des);

    for (int i = global_thread_idx; i < vec_len; i += grid_stride) {
        vec_des[i] = vec_src[i];
    }

    // handle tail part
    for (size_t i = vec_len * VEC_SIZE + global_thread_idx; i < length; i += grid_stride) {
        des[i] = src[i];
    }
}

__global__ __launch_bounds__(1024, 1) void all_gather_inter_ll_kernel(int8_t* q_ptr,
                                                                      int8_t* sym_buffer_ptr,
                                                                      int*    sym_signal_ptr,
                                                                      int32_t max_bs,
                                                                      int32_t msg_size,
                                                                      int32_t itemsize,
                                                                      int32_t world_size,
                                                                      int32_t rank,
                                                                      int32_t phases,
                                                                      bool    rdma_only)
{

    // Vectorize Optimization
    using vec_t        = int4;
    const int VEC_SIZE = sizeof(int4);

    const int num_sms = std::min(MAX_SMS, max_bs);

    const int sm_id   = blockIdx.x;
    const int warp_id = threadIdx.x / 32;
    const int lane_id = deep_ep::get_lane_id();

    const int dst_rank = warp_id;

    const int q_idx_base = sm_id * msg_size * itemsize;
    const int q_size     = max_bs * msg_size * itemsize;

    const int num_msg_per_warp     = msg_size * itemsize;
    const int num_vec_msg_per_warp = num_msg_per_warp / VEC_SIZE;
    int cnt = 0;

    if ((phases & ALL_GATHER_LL_SEND_PHASE) == 0)
        goto ALL_GATHER_LL_RECV;

    // Step 1. Write Q to buffer
    for (int q_idx = q_idx_base; q_idx < q_size; q_idx += num_sms * msg_size * itemsize) {

        if (dst_rank == rank) {
            int8_t*   q_ptr_for_write          = q_ptr + q_idx;
            vec_t*    vec_q_ptr_for_write      = reinterpret_cast<vec_t*>(q_ptr_for_write);
            const int buffer_idx               = q_idx + q_size * rank;
            int8_t*   buffer_ptr_for_write     = sym_buffer_ptr + buffer_idx;
            vec_t*    vec_buffer_ptr_for_write = reinterpret_cast<vec_t*>(buffer_ptr_for_write);

            UNROLLED_WARP_COPY(8,
                               lane_id,
                               num_vec_msg_per_warp,
                               vec_buffer_ptr_for_write,
                               vec_q_ptr_for_write,
                               deep_ep::ld_nc_global,
                               deep_ep::st_na_global);
        }
    }

    __syncthreads();

    // Step 2. Buffer Broadcast
    for (int q_idx = q_idx_base; q_idx < q_size; q_idx += num_sms * msg_size * itemsize) {
        if (dst_rank != rank) {
            const int slot_id = cnt * num_sms * world_size + sm_id * world_size + dst_rank;
            const int       buffer_idx           = q_idx + q_size * rank;
            const uintptr_t buffer_ptr_for_write = reinterpret_cast<uintptr_t>(sym_buffer_ptr + buffer_idx);
            const uintptr_t dst_buffer_p2p_ptr   = deep_ep::nvshmemi_get_p2p_ptr(buffer_ptr_for_write, rank, dst_rank);
            if (dst_buffer_p2p_ptr == 0 or rdma_only) {
                deep_ep::nvshmemi_ibgda_put_nbi_warp(
                    buffer_ptr_for_write, buffer_ptr_for_write, num_msg_per_warp, dst_rank, sm_id % 8, lane_id, 0);
            }
            else {
                vec_t* vec_buffer_ptr_for_write         = reinterpret_cast<vec_t*>(buffer_ptr_for_write);
                vec_t* vec_dst_buffer_p2p_ptr_for_write = reinterpret_cast<vec_t*>(dst_buffer_p2p_ptr);
                UNROLLED_WARP_COPY(8,
                                   lane_id,
                                   num_vec_msg_per_warp,
                                   vec_dst_buffer_p2p_ptr_for_write,
                                   vec_buffer_ptr_for_write,
                                   deep_ep::ld_nc_global,
                                   deep_ep::st_na_global);
            }
        }
        cnt += 1;
    }
    __syncwarp();

    // Step 3. Write Signal
    if (lane_id == 0) {
        const uintptr_t signal_ptr_for_write = reinterpret_cast<uintptr_t>(sym_signal_ptr + rank);
        const uintptr_t dst_signal_p2p_ptr =
            deep_ep::nvshmemi_get_p2p_ptr(reinterpret_cast<uintptr_t>(signal_ptr_for_write), rank, dst_rank);

        if (dst_signal_p2p_ptr == 0 or (dst_rank != rank and rdma_only))
            deep_ep::nvshmemi_ibgda_amo_nonfetch_add(sym_signal_ptr + rank, 1, dst_rank, sm_id % 8);
        else
            deep_ep::atomic_add_release_global(reinterpret_cast<int*>(dst_signal_p2p_ptr), 1);
    }
    __syncthreads();

    if ((phases & ALL_GATHER_LL_RECV_PHASE) == 0)
        return;

ALL_GATHER_LL_RECV:
    // Step 4. sync
    if (blockIdx.x == 0 and threadIdx.x < world_size) {

        while (deep_ep::ld_acquire_global(sym_signal_ptr + threadIdx.x) != num_sms)
            ;
        sym_signal_ptr[threadIdx.x] = 0;
    }
    return;
}

void all_gather_inter_ll(torch::Tensor q,
                         int8_t*       sym_buffer_ptr,
                         int*          sym_signal_ptr,
                         int32_t       max_bs,
                         int32_t       msg_size,
                         int32_t       itemsize,
                         int32_t       world_size,
                         int32_t       rank,
                         int           phase,
                         bool          rdma_only)
{

    int8_t* q_ptr = reinterpret_cast<int8_t*>(q.data_ptr());

    int num_sms   = std::min(128, max_bs);
    int num_warps = world_size;

    int grid_dim  = num_sms;
    int block_dim = num_warps * 32;

    auto stream = at::cuda::getCurrentCUDAStream();
    SETUP_LAUNCH_CONFIG(grid_dim, block_dim, stream);
    LAUNCH_KERNEL(&cfg,
                  all_gather_inter_ll_kernel,
                  q_ptr,
                  sym_buffer_ptr,
                  sym_signal_ptr,
                  max_bs,
                  msg_size,
                  itemsize,
                  world_size,
                  rank,
                  phase,
                  rdma_only);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return;
}

}  // namespace slime
