#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <stdexcept>

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <torch/torch.h>

#include "all_gather_inter_ll.h"
#include "ops/ibgda_device.cuh"
#include "ops/launch.cuh"
#include "ops/nvshmem_api.cuh"
#include "ops/utils.cuh"

namespace slime {

#define MAX_SMS                                                                                                        \
    int64_t                                                                                                            \
    {                                                                                                                  \
        128                                                                                                            \
    }

__global__ __launch_bounds__(1024, 1) void all_gather_inter_ll_kernel(int8_t* q_ptr,
                                                                      int8_t* sym_buffer_ptr,
                                                                      int*    sym_signal_ptr,
                                                                      int64_t max_bs,
                                                                      int64_t msg_size,
                                                                      int64_t itemsize,
                                                                      int64_t world_size,
                                                                      int64_t rank,
                                                                      int     phases,
                                                                      int64_t tag,
                                                                      bool    allow_nvlink)
{

    // Vectorize Optimization
    using vec_t        = int4;
    const int VEC_SIZE = sizeof(int4);

    const int num_sms = std::min(MAX_SMS, max_bs);

    const int sm_id   = blockIdx.x;
    const int warp_id = threadIdx.x / 32;
    const int lane_id = deep_ep::get_lane_id();

    const int dst_rank = warp_id;

    const int q_idx_base                  = sm_id * msg_size * itemsize;
    const int q_size                      = max_bs * msg_size * itemsize;
    const int buffer_size_per_concurrency = world_size * q_size;

    const int num_msg_per_warp     = msg_size * itemsize;
    const int num_vec_msg_per_warp = num_msg_per_warp / VEC_SIZE;

    if ((phases & ALL_GATHER_LL_SEND_PHASE) == 0)
        goto ALL_GATHER_LL_RECV;

    // Step 1. Write Q to buffer
    for (int q_idx = q_idx_base; q_idx < q_size; q_idx += num_sms * msg_size * itemsize) {

        if (dst_rank == rank) {
            int8_t*   q_ptr_for_write          = q_ptr + q_idx;
            vec_t*    vec_q_ptr_for_write      = reinterpret_cast<vec_t*>(q_ptr_for_write);
            const int buffer_idx               = tag * buffer_size_per_concurrency + q_idx + q_size * rank;
            int8_t*   buffer_ptr_for_write     = sym_buffer_ptr + buffer_idx;
            vec_t*    vec_buffer_ptr_for_write = reinterpret_cast<vec_t*>(buffer_ptr_for_write);

            UNROLLED_WARP_COPY(8,
                               lane_id,
                               num_vec_msg_per_warp,
                               vec_buffer_ptr_for_write,
                               vec_q_ptr_for_write,
                               deep_ep::ld_nc_global,
                               deep_ep::st_na_global);
        }
    }

    __syncthreads();

    // Step 2. Buffer Broadcast
    for (int q_idx = q_idx_base; q_idx < q_size; q_idx += num_sms * msg_size * itemsize) {
        if (dst_rank != rank) {
            const int       buffer_idx           = tag * buffer_size_per_concurrency + q_idx + q_size * rank;
            const uintptr_t buffer_ptr_for_write = reinterpret_cast<uintptr_t>(sym_buffer_ptr + buffer_idx);
            const uintptr_t dst_buffer_p2p_ptr   = deep_ep::nvshmemi_get_p2p_ptr(buffer_ptr_for_write, rank, dst_rank);
            if (dst_buffer_p2p_ptr == 0 or (not allow_nvlink)) {
                deep_ep::nvshmemi_ibgda_put_nbi_warp(
                    buffer_ptr_for_write, buffer_ptr_for_write, num_msg_per_warp, dst_rank, sm_id % 8, lane_id, 0);
            }
            else {
                vec_t* vec_buffer_ptr_for_write         = reinterpret_cast<vec_t*>(buffer_ptr_for_write);
                vec_t* vec_dst_buffer_p2p_ptr_for_write = reinterpret_cast<vec_t*>(dst_buffer_p2p_ptr);
                UNROLLED_WARP_COPY(8,
                                   lane_id,
                                   num_vec_msg_per_warp,
                                   vec_dst_buffer_p2p_ptr_for_write,
                                   vec_buffer_ptr_for_write,
                                   deep_ep::ld_nc_global,
                                   deep_ep::st_na_global);
            }
        }
    }
    __syncwarp();

    // Step 3. Write Signal
    if (lane_id == 0) {
        const uintptr_t signal_ptr_for_write = reinterpret_cast<uintptr_t>(sym_signal_ptr + tag * world_size + rank);
        const uintptr_t dst_signal_p2p_ptr =
            deep_ep::nvshmemi_get_p2p_ptr(reinterpret_cast<uintptr_t>(signal_ptr_for_write), rank, dst_rank);

        if (dst_signal_p2p_ptr == 0 or (dst_rank != rank and (not allow_nvlink))) {
            deep_ep::nvshmemi_ibgda_amo_nonfetch_add(sym_signal_ptr + tag * world_size + rank, 1, dst_rank, sm_id % 8);
        }
        else {
            deep_ep::atomic_add_release_global(reinterpret_cast<int*>(dst_signal_p2p_ptr), 1);
        }
    }
    __syncthreads();

    if ((phases & ALL_GATHER_LL_RECV_PHASE) == 0)
        return;

ALL_GATHER_LL_RECV:
    // Step 4. sync
    if (blockIdx.x == 0 and threadIdx.x < world_size) {
        const int* sym_signal_ptr_for_write = sym_signal_ptr + tag * world_size + threadIdx.x;
        while (deep_ep::ld_acquire_global(sym_signal_ptr_for_write) != num_sms)
            ;
        sym_signal_ptr[tag * world_size + threadIdx.x] = 0;
    }
    return;
}

void all_gather_inter_ll(torch::Tensor q,
                         int8_t*       sym_buffer_ptr,
                         int*          sym_signal_ptr,
                         int64_t       max_bs,
                         int64_t       msg_size,
                         int64_t       itemsize,
                         int64_t       world_size,
                         int64_t       rank,
                         int           phase,
                         int64_t       tag,
                         bool          allow_nvlink)
{

    int8_t* q_ptr = reinterpret_cast<int8_t*>(q.data_ptr());

    int num_sms   = std::min(int64_t{128}, max_bs);
    int num_warps = world_size;

    int grid_dim  = num_sms;
    int block_dim = num_warps * 32;

    auto stream = at::cuda::getCurrentCUDAStream();
    SETUP_LAUNCH_CONFIG(grid_dim, block_dim, stream);
    LAUNCH_KERNEL(&cfg,
                  all_gather_inter_ll_kernel,
                  q_ptr,
                  sym_buffer_ptr,
                  sym_signal_ptr,
                  max_bs,
                  msg_size,
                  itemsize,
                  world_size,
                  rank,
                  phase,
                  tag,
                  allow_nvlink);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return;
}

}  // namespace slime
