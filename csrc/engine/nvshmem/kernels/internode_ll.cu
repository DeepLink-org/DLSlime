#include "hip/hip_runtime.h"
#include "ibgda_device.cuh"
#include "utils/logging.h"
#include <cstdint>

namespace slime {

namespace internode {


__global__ void send_ll_kernel(int8_t* data, int8_t* buffer, int8_t* signal_buffer, size_t length, size_t msg_size_per_warp, size_t num_warps_per_sm, int rank, int dst_rank) {

    size_t alignment = msg_size_per_warp * num_warps_per_sm;
    size_t block_size = (length + alignment - 1) / alignment;
    size_t aligned_size = alignment * block_size;
    size_t warp_id = threadIdx.x / 32;
    size_t lane_id = threadIdx.x % 32;

    // Step 1. Data Copy
    size_t msg_size_per_thread = msg_size_per_warp / 32;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (int i = 0; i < msg_size_per_thread; ++i)
    {
        int idx = i + threadIdx.x * msg_size_per_thread + blockIdx.x * blockDim.x * msg_size_per_thread;
        if (idx < length)
            buffer[idx] = data[idx];
    }
    __syncthreads();

    // Step 2. Data Transformation
    uintptr_t buffer_ptr = reinterpret_cast<uintptr_t>(buffer) + warp_id * msg_size_per_warp + num_warps_per_sm * msg_size_per_warp * blockIdx.x;
    deep_ep::nvshmemi_ibgda_put_nbi_warp(buffer_ptr, buffer_ptr, msg_size_per_warp, dst_rank, 0, lane_id, 0);

    __syncthreads();

    // Step 3. Send Signal
    if (lane_id == 0) {
        auto signal_ptr = reinterpret_cast<uintptr_t>(signal_buffer) + blockIdx.x;
        deep_ep::nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int8_t*>(signal_ptr), 1, dst_rank, 0);
    }
}

__global__ void recv_ll_kernel(int8_t* data, int8_t* buffer, int8_t* signal_buffer, size_t length, size_t msg_size_per_warp, size_t num_warps_per_sm, int rank, int src_rank) {

    // Step 1. Data Copy
    size_t msg_size_per_thread = msg_size_per_warp / 32;
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x == 0) {
        while (__ldg(signal_buffer + blockIdx.x) != 32) {
        __threadfence_system();
        }
    }

    __syncthreads();

    for (int i = 0; i < msg_size_per_thread; ++i)
    {
        int idx = i + threadIdx.x * msg_size_per_thread + blockIdx.x * blockDim.x * msg_size_per_thread;
        if (idx < length)
            data[idx] = buffer[idx];
    }
    __syncthreads();

    signal_buffer[blockIdx.x] = 0;

}

void send_ll(int8_t* data, int8_t* buffer, int8_t* signal_buffer, size_t length, size_t msg_size_per_warp, size_t num_warps_per_sm, int rank, int dst_rank) {
    size_t alignment = msg_size_per_warp * num_warps_per_sm;
    size_t block_size = (length + alignment - 1) / alignment;
    size_t thread_size = num_warps_per_sm * 32;
    SLIME_LOG_INFO("block_size: " << block_size << ", thread_size: " << thread_size << ".");
    send_ll_kernel<<<block_size, thread_size>>>(data, buffer, signal_buffer, length, msg_size_per_warp, num_warps_per_sm, rank, dst_rank);
}

void recv_ll(int8_t* data, int8_t* buffer, int8_t* signal_buffer, size_t length, size_t msg_size_per_warp, size_t num_warps_per_sm, int rank, int src_rank) {
    size_t alignment = msg_size_per_warp * num_warps_per_sm;
    size_t block_size = (length + alignment - 1) / alignment;
    size_t thread_size = num_warps_per_sm * 32;
    SLIME_LOG_INFO("block_size: " << block_size << ", thread_size: " << thread_size << ".");
    recv_ll_kernel<<<block_size, thread_size>>>(data, buffer, signal_buffer, length, msg_size_per_warp, num_warps_per_sm, rank, src_rank);
}


}  // namespace internode
}  // namespace slime
